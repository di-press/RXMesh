#include "hip/hip_runtime.h"
#include "rxmesh/query.cuh"
#include "rxmesh/rxmesh_static.h"

#include "rxmesh/matrix/sparse_matrix.cuh"

using namespace rxmesh;


template <typename T, uint32_t blockThreads>
__global__ static void compute_area_matrix(const rxmesh::Context      context,
                                            rxmesh::VertexAttribute<T> boundaryVertices,
                                            rxmesh::SparseMatrix<T> AreaMatrix)
{

    auto vn_lambda = [&](EdgeHandle edge_id, VertexIterator& vv)
    {   
            
        if (boundaryVertices(vv[0], 0) == 1 && boundaryVertices(vv[1], 0) == 1){
            AreaMatrix(vv[0], vv[1]) = make_hipComplex(0,-0.25); // modify later
            AreaMatrix(vv[1], vv[0]) = make_hipComplex(0,0.25);
        }
        
    };

    auto block = cooperative_groups::this_thread_block();
    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::EV>(block, shrd_alloc, vn_lambda);
}

template <typename T>
__device__ __forceinline__ T
edge_cotan_weight(const rxmesh::VertexHandle&       p_id,
                  const rxmesh::VertexHandle&       r_id,
                  const rxmesh::VertexHandle&       q_id,
                  const rxmesh::VertexHandle&       s_id,
                  const rxmesh::VertexAttribute<T>& X)
{
    // Get the edge weight between the two vertices p-r where
    // q and s composes the diamond around p-r

    const vec3<T> p(X(p_id, 0), X(p_id, 1), X(p_id, 2));
    const vec3<T> r(X(r_id, 0), X(r_id, 1), X(r_id, 2));
    const vec3<T> q(X(q_id, 0), X(q_id, 1), X(q_id, 2));
    const vec3<T> s(X(s_id, 0), X(s_id, 1), X(s_id, 2));

    //cotans[(v1, v2)] =np.dot(e1, e2) / np.linalg.norm(np.cross(e1, e2))

    float weight = 0;
    if (q_id.is_valid())
        weight   += dot((p - q), (r - q)) / length(cross(p - q, r - q));
    if (s_id.is_valid())
        weight   += dot((p - s), (r - s)) / length(cross(p - s, r - s));
    weight /= 2;
    return std::max(0.f, weight);
}


template <typename T, uint32_t blockThreads>
__global__ static void compute_edge_weights_evd(const rxmesh::Context      context,
                                            rxmesh::VertexAttribute<T> coords,
                                            rxmesh::SparseMatrix<T>    A_mat)
{

    auto vn_lambda = [&](EdgeHandle edge_id, VertexIterator& vv) {
            T e_weight = 0;
            e_weight = edge_cotan_weight(vv[0], vv[2], vv[1], vv[3], coords);
        A_mat(vv[0], vv[2]) = e_weight;
        A_mat(vv[2], vv[0]) = e_weight;

        //A_mat(vv[0], vv[2]) = 1;
        //A_mat(vv[2], vv[0]) = 1;
        
    };

    auto                block = cooperative_groups::this_thread_block();
    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::EVDiamond>(block, shrd_alloc, vn_lambda);
}


__global__ static void calculate_Ld_matrix(
    const rxmesh::Context   context,
    rxmesh::SparseMatrix<T> weight_mat,  // [num_coord, num_coord]
    rxmesh::SparseMatrix<T> Ld // [num_coord, num_coord]
)

{
    auto init_lambda = [&](VertexHandle v_id, VertexIterator& vv) {

        L(v_id, v_id) =  make_hipComplex(0,0);

        for (int nei_index = 0; nei_index < vv.size(); nei_index++)
            L(v_id, vv[nei_index]) =  make_hipComplex(0,0);

            for (int nei_index = 0; nei_index < vv.size(); nei_index++) 
            {
                L(v_id, v_id) +=  make_hipComplex(weight_mat(v_id, vv[nei_index]), weight_mat(v_id, vv[nei_index]));
                L(v_id, vv[nei_index]) -= make_hipComplex(weight_mat(v_id, vv[nei_index]), 0);
            }


    };

    auto                block = cooperative_groups::this_thread_block();
    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::VV>(block, shrd_alloc, init_lambda);
}

int main(int argc, char** argv)
{
    Log::init();

    const uint32_t device_id = 0;
    cuda_query(device_id);

    RXMeshStatic rx(STRINGIFY(INPUT_DIR) "bunnyhead.obj");

    SparseMatrix<hipComplex> Ld(rx); //complex V x V

    SparseMatrix<hipComplex> A(rx); // 2V x 2V

    auto boundaryVertices = *rx.add_vertex_attribute<int>("boundaryVertices", 1);

    rx.get_boundary_vertices(boundaryVertices); // 0 or 1 value for boundary vertex

    // identify boundary edge (vv query)
    // v1 is central; v2 is on boundary 






#if USE_POLYSCOPE
    polyscope::show();
#endif
}